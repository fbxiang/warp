#include "warp.h"
#include <hipsparse.h>

namespace wp {

static hipsparseHandle_t g_cusparse_handle;


bool init_cusparse() {
  hipsparseStatus_t status = hipsparseCreate(&g_cusparse_handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    fprintf(stderr, "%s\n", hipsparseGetErrorString(status));
    return false;
  }
  hipsparseSetStream(g_cusparse_handle, (hipStream_t)cuda_stream_get_current());
  return true;
}
void destroy_cusparse() { hipsparseDestroy(g_cusparse_handle); }
void *get_cusparse_handle() { return (void *)g_cusparse_handle; }

} // namespace wp

// -------------------- begin helper functions --------------------

template <typename T> struct CSRMatrix {
  int m{};
  int nnz{};
  int *offsets{};
  int *columns{};
  T *values{};
  hipsparseMatDescr_t matDescr{};
  hipsparseSpMatDescr_t spMatDescr{};

  csric02Info_t icInfo{};
  csrilu02Info_t iluInfo{};
  hipsparseSpSVDescr_t svInfo{};
};

template <typename T> struct DenseVector {
  int m{};
  T *values{};
  hipsparseDnVecDescr_t vecDescr{};
};

uint64_t dense_vector_create_device(int m, float *d_values) {
  DenseVector<float> *vec = new DenseVector<float>;
  vec->m = m;
  vec->values = d_values;
  hipsparseCreateDnVec(&vec->vecDescr, m, d_values, HIP_R_32F);
  return (uint64_t)vec;
}

void dense_vector_destroy_device(uint64_t id) {
  DenseVector<void> *vec = (DenseVector<void> *)id;
  hipsparseDestroyDnVec(vec->vecDescr);
  delete vec;
}

uint64_t csr_create_device(int m, int nnz, int *d_offsets, int *d_columns, float *d_values, int fillmode,
                           int diagtype) {
  CSRMatrix<float> *mat = new CSRMatrix<float>;
  mat->m = m;
  mat->nnz = nnz;
  mat->offsets = d_offsets;
  mat->columns = d_columns;
  mat->values = d_values;

  hipsparseCreateMatDescr(&mat->matDescr);
  hipsparseSetMatIndexBase(mat->matDescr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(mat->matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);

  hipsparseFillMode_t fillmode_ = (hipsparseFillMode_t)fillmode;
  hipsparseDiagType_t diagtype_ = (hipsparseDiagType_t)diagtype;

  hipsparseSetMatFillMode(mat->matDescr, fillmode_);
  hipsparseSetMatDiagType(mat->matDescr, diagtype_);

  hipsparseCreateCsr(&mat->spMatDescr, m, m, nnz, d_offsets, d_columns, d_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
  hipsparseSpMatSetAttribute(mat->spMatDescr, HIPSPARSE_SPMAT_FILL_MODE, &fillmode_, sizeof(hipsparseFillMode_t));
  hipsparseSpMatSetAttribute(mat->spMatDescr, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype_, sizeof(hipsparseDiagType_t));

  return (uint64_t)mat;
}

void csr_destroy_device(uint64_t id) {
  CSRMatrix<void> *mat = (CSRMatrix<void> *)id;
  hipsparseDestroyMatDescr(mat->matDescr);
  hipsparseDestroySpMat(mat->spMatDescr);
  if (mat->icInfo) {
    hipsparseDestroyCsric02Info(mat->icInfo);
  }
  if (mat->iluInfo) {
    hipsparseDestroyCsrilu02Info(mat->iluInfo);
  }
  delete mat;
}

// incomplete cholesky
template <typename T>
inline hipsparseStatus_t cusparseXcsric02_bufferSize(hipsparseHandle_t handle, int m, int nnz,
                                                    const hipsparseMatDescr_t descrA, T *csrSortedValA,
                                                    const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                    csric02Info_t info, int *pBufferSizeInBytes) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsric02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                       pBufferSizeInBytes);
  } else {
    return hipsparseDcsric02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                       pBufferSizeInBytes);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsric02_analysis(hipsparseHandle_t handle, int m, int nnz,
                                                  const hipsparseMatDescr_t descrA, const T *csrSortedValA,
                                                  const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                  csric02Info_t info, hipsparseSolvePolicy_t policy, void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsric02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                     policy, pBuffer);
  } else {
    return hipsparseDcsric02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                     policy, pBuffer);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsric02(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA,
                                         T *csrSortedValA_valM, const int *csrSortedRowPtrA,
                                         const int *csrSortedColIndA, csric02Info_t info, hipsparseSolvePolicy_t policy,
                                         void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsric02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                            policy, pBuffer);
  } else {
    return hipsparseDcsric02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                            policy, pBuffer);
  }
}

// incomplete LU
template <typename T>
inline hipsparseStatus_t cusparseXcsrilu02_bufferSize(hipsparseHandle_t handle, int m, int nnz,
                                                     const hipsparseMatDescr_t descrA, T *csrSortedValA,
                                                     const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                     csrilu02Info_t info, int *pBufferSizeInBytes) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsrilu02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                        pBufferSizeInBytes);
  } else {
    return hipsparseDcsrilu02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                        pBufferSizeInBytes);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsrilu02_analysis(hipsparseHandle_t handle, int m, int nnz,
                                                   const hipsparseMatDescr_t descrA, const T *csrSortedValA,
                                                   const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                   csrilu02Info_t info, hipsparseSolvePolicy_t policy, void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsrilu02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                      policy, pBuffer);
  } else {
    return hipsparseDcsrilu02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                      policy, pBuffer);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsrilu02(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA,
                                          T *csrSortedValA_valM, const int *csrSortedRowPtrA,
                                          const int *csrSortedColIndA, csrilu02Info_t info,
                                          hipsparseSolvePolicy_t policy, void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsrilu02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                             policy, pBuffer);
  } else {
    return hipsparseDcsrilu02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                             policy, pBuffer);
  }
}

// -------------------- end helper functions --------------------

template <typename ValueType = float> static int _csr_ichol_device_buffer_size(uint64_t id) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *mat = (CSRMatrix<ValueType> *)id;
  int bufferSize{0};
  if (!mat->icInfo) {
    hipsparseCreateCsric02Info(&mat->icInfo);
  }
  cusparseXcsric02_bufferSize<ValueType>(cusparse_handle, mat->m, mat->nnz, mat->matDescr, mat->values, mat->offsets,
                                         mat->columns, mat->icInfo, &bufferSize);
  return bufferSize;
}

template <typename ValueType = float> static void _csr_ichol_device(uint64_t matA, ValueType *L_values, void *buffer) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)matA;

  if (!A->icInfo) {
    hipsparseCreateCsric02Info(&A->icInfo);
  }

  // L = A
  hipMemcpyAsync(L_values, A->values, A->nnz * sizeof(ValueType), hipMemcpyDeviceToDevice);
  cusparseXcsric02_analysis<ValueType>(cusparse_handle, A->m, A->nnz, A->matDescr, L_values, A->offsets, A->columns,
                                       A->icInfo, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
  // TODO
  // int structural_zero; hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &structural_zero);
  cusparseXcsric02<ValueType>(cusparse_handle, A->m, A->nnz, A->matDescr, L_values, A->offsets, A->columns, A->icInfo,
                              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
  // TODO
  // int numerical_zero; hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &numerical_zero);
}

template <typename ValueType = float> static int _csr_ilu_device_buffer_size(uint64_t id) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *mat = (CSRMatrix<ValueType> *)id;
  int bufferSize{0};
  if (!mat->iluInfo) {
    hipsparseCreateCsrilu02Info(&mat->iluInfo);
  }
  cusparseXcsrilu02_bufferSize<ValueType>(cusparse_handle, mat->m, mat->nnz, mat->matDescr, mat->values, mat->offsets,
                                          mat->columns, mat->iluInfo, &bufferSize);
  return bufferSize;
}

template <typename ValueType = float> static void _csr_ilu_device(uint64_t matA, ValueType *LU_values, void *buffer) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)matA;

  if (!A->iluInfo) {
    hipsparseCreateCsrilu02Info(&A->iluInfo);
  }

  hipMemcpyAsync(LU_values, A->values, A->nnz * sizeof(ValueType), hipMemcpyDeviceToDevice);
  cusparseXcsrilu02_analysis<ValueType>(cusparse_handle, A->m, A->nnz, A->matDescr, LU_values, A->offsets, A->columns,
                                        A->iluInfo, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
  // TODO
  // int structural_zero; hipsparseXcsrilu02_zeroPivot(cusparse_handle, infoM, &structural_zero);
  cusparseXcsrilu02<ValueType>(cusparse_handle, A->m, A->nnz, A->matDescr, LU_values, A->offsets, A->columns,
                               A->iluInfo, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
  // TODO
  // int numerical_zero; hipsparseXcsrilu02_zeroPivot(cusparse_handle, infoM, &numerical_zero);
}

template <typename ValueType = float>
static int _csr_mv_device_buffer_size(uint64_t idA, uint64_t idX, uint64_t idY, ValueType alpha, ValueType beta) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)idA;
  DenseVector<ValueType> *X = (DenseVector<ValueType> *)idX;
  DenseVector<ValueType> *Y = (DenseVector<ValueType> *)idY;
  size_t bufferSize{};
  constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;
  hipsparseSpMV_bufferSize(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, A->spMatDescr, X->vecDescr, &beta,
                          Y->vecDescr, valueType, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
  return (int)bufferSize;
}

template <typename ValueType = float>
static void _csr_mv_device(uint64_t idA, uint64_t idX, uint64_t idY, ValueType alpha, ValueType beta, void *buffer) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)idA;
  DenseVector<ValueType> *X = (DenseVector<ValueType> *)idX;
  DenseVector<ValueType> *Y = (DenseVector<ValueType> *)idY;

  constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;
  hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, A->spMatDescr, X->vecDescr, &beta,
               Y->vecDescr, valueType, HIPSPARSE_SPMV_ALG_DEFAULT, buffer);
}

int csr_ichol_device_buffer_size(uint64_t id) { return _csr_ichol_device_buffer_size<float>(id); }
void csr_ichol_device(uint64_t id, float *L_values, void *buffer) { _csr_ichol_device<float>(id, L_values, buffer); }
int csr_ilu_device_buffer_size(uint64_t id) { return _csr_ilu_device_buffer_size<float>(id); }
void csr_ilu_device(uint64_t id, float *LU_values, void *buffer) { _csr_ilu_device<float>(id, LU_values, buffer); }

int csr_mv_device_buffer_size(uint64_t idA, uint64_t idX, uint64_t idY, float alpha, float beta) {
  return _csr_mv_device_buffer_size<float>(idA, idX, idY, alpha, beta);
}
void csr_mv_device(uint64_t idA, uint64_t idX, uint64_t idY, float alpha, float beta, void *buffer) {
  _csr_mv_device<float>(idA, idX, idY, alpha, beta, buffer);
}

template <typename ValueType>
static int _csr_sv_device_buffer_size(uint64_t idA, uint64_t idX, uint64_t idY, float alpha, int op) {
  hipsparseHandle_t handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)idA;
  DenseVector<ValueType> *X = (DenseVector<ValueType> *)idX;
  DenseVector<ValueType> *Y = (DenseVector<ValueType> *)idY;

  constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;

  if (!A->svInfo) {
    hipsparseSpSV_createDescr(&A->svInfo);
  }

  size_t bufferSize{0};
  hipsparseSpSV_bufferSize(handle, (hipsparseOperation_t)op, &alpha, A->spMatDescr, X->vecDescr, Y->vecDescr, valueType,
                          HIPSPARSE_SPSV_ALG_DEFAULT, A->svInfo, &bufferSize);

  return (int)bufferSize;
}

template <typename ValueType>
static void _csr_sv_device(uint64_t idA, uint64_t idX, uint64_t idY, float alpha, int op, void *buffer) {
  hipsparseHandle_t handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)idA;
  DenseVector<ValueType> *X = (DenseVector<ValueType> *)idX;
  DenseVector<ValueType> *Y = (DenseVector<ValueType> *)idY;

  constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;

  if (A->svInfo) {
    hipsparseSpSV_createDescr(&A->svInfo);
  }

  hipsparseSpSV_analysis(handle, (hipsparseOperation_t)op, &alpha, A->spMatDescr, X->vecDescr, Y->vecDescr, valueType,
                        HIPSPARSE_SPSV_ALG_DEFAULT, A->svInfo, buffer);
  hipsparseSpSV_solve(handle, (hipsparseOperation_t)op, &alpha, A->spMatDescr, X->vecDescr, Y->vecDescr, valueType,
                     HIPSPARSE_SPSV_ALG_DEFAULT, A->svInfo);
}

int csr_sv_device_buffer_size(uint64_t idA, uint64_t idX, uint64_t idY, float alpha, int op) {
  return _csr_sv_device_buffer_size<float>(idA, idX, idY, alpha, op);
}

void csr_sv_device(uint64_t idA, uint64_t idX, uint64_t idY, float alpha, int op, void *buffer) {
  _csr_sv_device<float>(idA, idX, idY, alpha, op, buffer);
}
