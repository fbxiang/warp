#include "hip/hip_runtime.h"
#include "warp.h"
#include <hipblas.h>
#include <hipsparse.h>

namespace wp {

static hipblasHandle_t g_cublas_handle;
static hipsparseHandle_t g_cusparse_handle;

bool init_cublas() {
  hipblasStatus_t status = hipblasCreate(&g_cublas_handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUBLAS error: %d\n", status);
    return false;
  }
  hipblasSetStream(g_cublas_handle, (hipStream_t)cuda_stream_get_current());
  return true;
}
void destroy_cublas() { hipblasDestroy(g_cublas_handle); }
void *get_cublas_handle() { return (void *)g_cublas_handle; }

bool init_cusparse() {
  hipsparseStatus_t status = hipsparseCreate(&g_cusparse_handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    fprintf(stderr, "%s\n", hipsparseGetErrorString(status));
    return false;
  }
  hipsparseSetStream(g_cusparse_handle, (hipStream_t)cuda_stream_get_current());
  return true;
}
void destroy_cusparse() { hipsparseDestroy(g_cusparse_handle); }
void *get_cusparse_handle() { return (void *)g_cusparse_handle; }

} // namespace wp

// -------------------- begin helper functions --------------------

template <typename T> struct CSRMatrix {
  int m{};
  int nnz{};
  int *offsets{};
  int *columns{};
  T *values{};
  hipsparseMatDescr_t matDescr{};
  hipsparseSpMatDescr_t spMatDescr{};
  csric02Info_t icInfo{};
  csrilu02Info_t iluInfo{};
};

template<typename T> struct DenseVector {
  int m{};
  T *values{};
  hipsparseDnVecDescr_t vecDescr{};
};

uint64_t dense_vector_create_device(int m, float *d_values) {
  DenseVector<float> *vec = new DenseVector<float>;
  vec->m = m;
  vec->values = d_values;
  hipsparseCreateDnVec(&vec->vecDescr, m, d_values, HIP_R_32F);
  return (uint64_t) vec;
}

void dense_vector_destroy_device(uint64_t id) {
  DenseVector<void> *vec = (DenseVector<void> *)id;
  hipsparseDestroyDnVec(vec->vecDescr);
  delete vec;
}

uint64_t csr_create_device(int m, int nnz, int *d_offsets, int *d_columns, float *d_values) {
  CSRMatrix<float> *mat = new CSRMatrix<float>;
  mat->m = m;
  mat->nnz = nnz;
  mat->offsets = d_offsets;
  mat->columns = d_columns;
  mat->values = d_values;

  hipsparseCreateMatDescr(&mat->matDescr);
  hipsparseSetMatIndexBase(mat->matDescr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(mat->matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatFillMode(mat->matDescr, HIPSPARSE_FILL_MODE_LOWER);
  hipsparseSetMatDiagType(mat->matDescr, HIPSPARSE_DIAG_TYPE_NON_UNIT);

  hipsparseCreateCsr(&mat->spMatDescr, m, m, nnz, d_offsets, d_columns, d_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

  return (uint64_t)mat;
}

void csr_destroy_device(uint64_t id) {
  CSRMatrix<void> *mat = (CSRMatrix<void> *)id;
  hipsparseDestroyMatDescr(mat->matDescr);
  hipsparseDestroySpMat(mat->spMatDescr);
  if (mat->icInfo) {
    hipsparseDestroyCsric02Info(mat->icInfo);
  }
  if (mat->iluInfo) {
    hipsparseDestroyCsrilu02Info(mat->iluInfo);
  }
  delete mat;
}

// incomplete cholesky
template <typename T>
inline hipsparseStatus_t cusparseXcsric02_bufferSize(hipsparseHandle_t handle, int m, int nnz,
                                                    const hipsparseMatDescr_t descrA, T *csrSortedValA,
                                                    const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                    csric02Info_t info, int *pBufferSizeInBytes) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsric02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                       pBufferSizeInBytes);
  } else {
    return hipsparseDcsric02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                       pBufferSizeInBytes);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsric02_analysis(hipsparseHandle_t handle, int m, int nnz,
                                                  const hipsparseMatDescr_t descrA, const T *csrSortedValA,
                                                  const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                  csric02Info_t info, hipsparseSolvePolicy_t policy, void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsric02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                     policy, pBuffer);
  } else {
    return hipsparseDcsric02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                     policy, pBuffer);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsric02(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA,
                                         T *csrSortedValA_valM, const int *csrSortedRowPtrA,
                                         const int *csrSortedColIndA, csric02Info_t info, hipsparseSolvePolicy_t policy,
                                         void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsric02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                            policy, pBuffer);
  } else {
    return hipsparseDcsric02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                            policy, pBuffer);
  }
}

// incomplete LU
template <typename T>
inline hipsparseStatus_t cusparseXcsrilu02_bufferSize(hipsparseHandle_t handle, int m, int nnz,
                                                     const hipsparseMatDescr_t descrA, T *csrSortedValA,
                                                     const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                     csrilu02Info_t info, int *pBufferSizeInBytes) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsrilu02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                        pBufferSizeInBytes);
  } else {
    return hipsparseDcsrilu02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                        pBufferSizeInBytes);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsrilu02_analysis(hipsparseHandle_t handle, int m, int nnz,
                                                   const hipsparseMatDescr_t descrA, const T *csrSortedValA,
                                                   const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                   csrilu02Info_t info, hipsparseSolvePolicy_t policy, void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsrilu02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                      policy, pBuffer);
  } else {
    return hipsparseDcsrilu02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                      policy, pBuffer);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsrilu02(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA,
                                          T *csrSortedValA_valM, const int *csrSortedRowPtrA,
                                          const int *csrSortedColIndA, csrilu02Info_t info,
                                          hipsparseSolvePolicy_t policy, void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsrilu02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                             policy, pBuffer);
  } else {
    return hipsparseDcsrilu02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                             policy, pBuffer);
  }
}

// -------------------- end helper functions --------------------

template <typename ValueType = float> static int _csr_ichol_device_buffer_size(uint64_t id) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *mat = (CSRMatrix<ValueType> *)id;
  int bufferSize{0};
  if (!mat->icInfo) {
    hipsparseCreateCsric02Info(&mat->icInfo);
  }
  cusparseXcsric02_bufferSize<ValueType>(cusparse_handle, mat->m, mat->nnz, mat->matDescr, mat->values, mat->offsets,
                                         mat->columns, mat->icInfo, &bufferSize);
  return bufferSize;
}

template <typename ValueType = float> static void _csr_ichol_device(uint64_t matA, ValueType *L_values, void *buffer) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)matA;

  if (!A->icInfo) {
    hipsparseCreateCsric02Info(&A->icInfo);
  }

  // L = A
  hipMemcpyAsync(L_values, A->values, A->nnz * sizeof(ValueType), hipMemcpyDeviceToDevice);
  cusparseXcsric02_analysis<ValueType>(cusparse_handle, A->m, A->nnz, A->matDescr, L_values, A->offsets, A->columns,
                                       A->icInfo, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
  // TODO
  // int structural_zero; hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &structural_zero);
  cusparseXcsric02<ValueType>(cusparse_handle, A->m, A->nnz, A->matDescr, L_values, A->offsets, A->columns, A->icInfo,
                              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
  // TODO
  // int numerical_zero; hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &numerical_zero);
}

template <typename ValueType = float> static int _csr_ilu_device_buffer_size(uint64_t id) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *mat = (CSRMatrix<ValueType> *)id;
  int bufferSize{0};
  if (!mat->iluInfo) {
    hipsparseCreateCsrilu02Info(&mat->iluInfo);
  }
  cusparseXcsrilu02_bufferSize<ValueType>(cusparse_handle, mat->m, mat->nnz, mat->matDescr, mat->values, mat->offsets,
                                          mat->columns, mat->iluInfo, &bufferSize);
  return bufferSize;
}

template <typename ValueType = float> static void _csr_ilu_device(uint64_t matA, ValueType *LU_values, void *buffer) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)matA;

  if (!A->iluInfo) {
    hipsparseCreateCsrilu02Info(&A->iluInfo);
  }

  hipMemcpyAsync(LU_values, A->values, A->nnz * sizeof(ValueType), hipMemcpyDeviceToDevice);
  cusparseXcsrilu02_analysis<ValueType>(cusparse_handle, A->m, A->nnz, A->matDescr, LU_values, A->offsets, A->columns,
                                        A->iluInfo, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
  // TODO
  // int structural_zero; hipsparseXcsrilu02_zeroPivot(cusparse_handle, infoM, &structural_zero);
  cusparseXcsrilu02<ValueType>(cusparse_handle, A->m, A->nnz, A->matDescr, LU_values, A->offsets, A->columns, A->iluInfo,
                               HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
  // TODO
  // int numerical_zero; hipsparseXcsrilu02_zeroPivot(cusparse_handle, infoM, &numerical_zero);
}

template <typename ValueType = float>
static int _csr_mv_device_buffer_size(uint64_t idA, uint64_t idX, uint64_t idY, ValueType alpha, ValueType beta) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)idA;
  DenseVector<ValueType> *X = (DenseVector<ValueType> *)idX;
  DenseVector<ValueType> *Y = (DenseVector<ValueType> *)idY;
  size_t bufferSize{};
  constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;
  hipsparseSpMV_bufferSize(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, A->spMatDescr,
                          X->vecDescr, &beta, Y->vecDescr, valueType,
                          HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
  return (int)bufferSize;
}

template <typename ValueType = float>
static void _csr_mv_device(uint64_t idA, uint64_t idX, uint64_t idY, ValueType alpha, ValueType beta, void *buffer) {
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  CSRMatrix<ValueType> *A = (CSRMatrix<ValueType> *)idA;
  DenseVector<ValueType> *X = (DenseVector<ValueType> *)idX;
  DenseVector<ValueType> *Y = (DenseVector<ValueType> *)idY;

  constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;
  hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, A->spMatDescr,
               X->vecDescr, &beta, Y->vecDescr, valueType,
               HIPSPARSE_SPMV_ALG_DEFAULT, buffer);
}

// template <typename ValueType = float>
// static int _csr_mv_device_buffer_size(int m, int nnz, int *offsets, int *columns, ValueType *values, ValueType *x,
//                                       ValueType *y, ValueType alpha, ValueType beta) {
//   hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();

//   constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;
//   hipsparseSpMatDescr_t matA{};
//   hipsparseDnVecDescr_t vecX{};
//   hipsparseDnVecDescr_t vecY{};
//   hipsparseCreateCsr(&matA, m, m, nnz, offsets, columns, values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
//                     HIPSPARSE_INDEX_BASE_ZERO, valueType);
//   hipsparseCreateDnVec(&vecX, m, x, valueType);
//   hipsparseCreateDnVec(&vecY, m, y, valueType);

//   size_t bufferSize{};

//   hipsparseSpMV_bufferSize(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, valueType,
//                           HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);

//   hipsparseDestroySpMat(matA);
//   hipsparseDestroyDnVec(vecX);
//   hipsparseDestroyDnVec(vecY);
//   return (int)bufferSize;
// }

// template <typename ValueType = float>
// static void _csr_mv_device(int m, int nnz, int *offsets, int *columns, ValueType *values, ValueType *x, ValueType *y,
//                            ValueType alpha, ValueType beta, void *buffer) {
//   hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();

//   constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;
//   hipsparseSpMatDescr_t matA{};
//   hipsparseDnVecDescr_t vecX{};
//   hipsparseDnVecDescr_t vecY{};
//   hipsparseCreateCsr(&matA, m, m, nnz, offsets, columns, values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
//                     HIPSPARSE_INDEX_BASE_ZERO, valueType);
//   hipsparseCreateDnVec(&vecX, m, x, valueType);
//   hipsparseCreateDnVec(&vecY, m, y, valueType);

//   hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, valueType,
//                HIPSPARSE_SPMV_ALG_DEFAULT, buffer);
//   hipsparseDestroySpMat(matA);
//   hipsparseDestroyDnVec(vecX);
//   hipsparseDestroyDnVec(vecY);
// }

int csr_ichol_device_buffer_size(uint64_t id) { return _csr_ichol_device_buffer_size<float>(id); }
void csr_ichol_device(uint64_t id, float *L_values, void *buffer) { _csr_ichol_device<float>(id, L_values, buffer); }
int csr_ilu_device_buffer_size(uint64_t id) { return _csr_ilu_device_buffer_size<float>(id); }
void csr_ilu_device(uint64_t id, float *LU_values, void *buffer) { _csr_ilu_device<float>(id, LU_values, buffer); }

int csr_mv_device_buffer_size(uint64_t idA, uint64_t idX, uint64_t idY, float alpha, float beta) {
  return _csr_mv_device_buffer_size<float>(idA, idX, idY, alpha, beta);
}
void csr_mv_device(uint64_t idA, uint64_t idX, uint64_t idY, float alpha, float beta, void *buffer) {
  _csr_mv_device<float>(idA, idX, idY, alpha, beta, buffer);
}

// /** Ax = b */
// template <typename ValueType = float>
// static void csr_ic_cg(int m, int nnz, int *offsets, int *columns, ValueType *values, // A
//                       ValueType *p_b,                                                // RHS b
//                       ValueType *p_x,                                                // initial x
//                       ValueType *p_r,                                                // residual
//                       int max_iterations, ValueType rtol) {
//   const ValueType minus_one = -1;
//   const ValueType one = 1;
//   const ValueType zero = 0;
//   hipsparseFillMode_t fill_lower = HIPSPARSE_FILL_MODE_LOWER;
//   hipsparseDiagType_t diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

//   // set up handles
//   hipStream_t stream = (hipStream_t)cuda_stream_get_current();
//   hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
//   hipblasHandle_t cublas_handle = (hipblasHandle_t)wp::get_cublas_handle();
//   hipsparseSetStream(cusparse_handle, stream);
//   hipblasSetStream(cublas_handle, stream);

//   // set up data type
//   static_assert(std::is_same<ValueType, float>::value || std::is_same<ValueType, double>::value,
//                 "invalid data type for csr_cg");
//   constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;

//   // set up descriptors
//   hipsparseSpMatDescr_t matA{};
//   hipsparseSpMatDescr_t matL{};

//   hipsparseCreateCsr(&matA, m, m, nnz, offsets, columns, values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
//                     HIPSPARSE_INDEX_BASE_ZERO, valueType);

//   ValueType *L_values{};
//   hipMallocAsync(&L_values, nnz * sizeof(ValueType), stream);
//   hipMemcpyAsync(L_values, values, nnz * sizeof(ValueType), hipMemcpyDeviceToDevice);

//   hipsparseCreateCsr(&matL, m, m, nnz, offsets, columns, L_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
//                     HIPSPARSE_INDEX_BASE_ZERO, valueType);
//   hipsparseSpMatSetAttribute(matL, HIPSPARSE_SPMAT_FILL_MODE, &fill_lower, sizeof(fill_lower));
//   hipsparseSpMatSetAttribute(matL, HIPSPARSE_SPMAT_DIAG_TYPE, &diag_non_unit, sizeof(diag_non_unit));

//   // incomplete Cholesky
//   hipsparseMatDescr_t matM{};
//   csric02Info_t infoM{};
//   int bufferSizeIC{0};
//   void *bufferIC;
//   hipsparseCreateMatDescr(&matM);
//   hipsparseSetMatIndexBase(matM, HIPSPARSE_INDEX_BASE_ZERO);
//   hipsparseSetMatType(matM, HIPSPARSE_MATRIX_TYPE_GENERAL);
//   hipsparseSetMatFillMode(matM, HIPSPARSE_FILL_MODE_LOWER);
//   hipsparseSetMatDiagType(matM, HIPSPARSE_DIAG_TYPE_NON_UNIT);
//   hipsparseCreateCsric02Info(&infoM);

//   if constexpr (std::is_same<ValueType, float>::value) {
//     hipsparseScsric02_bufferSize(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM, &bufferSizeIC);
//   } else {
//     hipsparseDcsric02_bufferSize(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM, &bufferSizeIC);
//   }
//   hipMallocAsync(&bufferIC, bufferSizeIC, stream); // TODO preallocate

//   if constexpr (std::is_same<ValueType, float>::value) {
//     hipsparseScsric02_analysis(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM,
//                               HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferIC);
//   } else {
//     hipsparseDcsric02_analysis(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM,
//                               HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferIC);
//   }

//   // TODO
//   // int structural_zero;
//   // hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &structural_zero);

//   if constexpr (std::is_same<ValueType, float>::value) {
//     hipsparseScsric02(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM,
//     HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
//                      bufferIC);
//   } else {
//     hipsparseDcsric02(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM,
//     HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
//                      bufferIC);
//   }

//   // TODO
//   // int numerical_zero;
//   // hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &numerical_zero);

//   hipsparseDestroyCsric02Info(infoM);
//   hipsparseDestroyMatDescr(matM);
//   hipFreeAsync(bufferIC, stream);

//   //// CG

//   // set up
//   hipsparseDnVecDescr_t vecB{};
//   hipsparseDnVecDescr_t vecR{};
//   hipsparseDnVecDescr_t vecX{};
//   hipsparseDnVecDescr_t vecTmp{};
//   hipsparseDnVecDescr_t vecRaux{};
//   hipsparseDnVecDescr_t vecP{};
//   hipsparseDnVecDescr_t vecT{};

//   ValueType *p_tmp{};
//   hipMallocAsync(&p_tmp, m * sizeof(ValueType), stream);

//   hipsparseCreateDnVec(&vecTmp, m, p_tmp, valueType);
//   ValueType *p_raux{};
//   hipMallocAsync(&p_raux, m * sizeof(ValueType), stream);
//   hipsparseCreateDnVec(&vecRaux, m, p_raux, valueType);

//   ValueType *p_p{};
//   hipMallocAsync(&p_p, m * sizeof(ValueType), stream);
//   hipsparseCreateDnVec(&vecP, m, p_p, valueType);

//   ValueType *p_t{};
//   hipMallocAsync(&p_t, m * sizeof(ValueType), stream);
//   hipsparseCreateDnVec(&vecT, m, p_t, valueType);

//   hipsparseCreateDnVec(&vecB, m, p_b, valueType);
//   hipsparseCreateDnVec(&vecR, m, p_r, valueType);
//   hipsparseCreateDnVec(&vecX, m, p_x, valueType);

//   size_t bufferSizeMV{};
//   void *bufferMV{};
//   // allocate memory
//   hipsparseSpMV_bufferSize(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one, matA, vecX, &one, vecB,
//                           valueType, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSizeMV);
//   hipMallocAsync(&bufferMV, bufferSizeMV, stream); // TODO preallocate and reuse

//   // FIXME check bufferSizeMV for all vecs

//   // r0 = b - A@x0
//   hipMemcpyAsync(p_r, p_b, m * sizeof(ValueType), hipMemcpyDeviceToDevice, stream);
//   hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one, matA, vecX, &one, vecR, valueType,
//                HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV);

//   // tmp = L^-T r
//   size_t bufferSizeL, bufferSizeLT;
//   void *bufferL, *bufferLT;
//   hipsparseSpSVDescr_t spsvDescrL, spsvDescrLT;
//   hipsparseSpSV_createDescr(&spsvDescrLT);

//   hipsparseSpSV_bufferSize(cusparse_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, vecR, vecTmp, valueType,
//                           HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT, &bufferSizeLT);
//   hipMallocAsync(&bufferLT, bufferSizeLT, stream);
//   hipsparseSpSV_analysis(cusparse_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, vecR, vecTmp, valueType,
//                         HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT, bufferLT);
//   hipMemsetAsync(p_tmp, 0x0, m * sizeof(ValueType), stream);
//   hipsparseSpSV_solve(cusparse_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, vecR, vecTmp, valueType,
//                      HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT);

//   // raux = L^-1 L^-T r
//   hipsparseSpSV_createDescr(&spsvDescrL);

//   hipsparseSpSV_bufferSize(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, vecTmp, vecRaux,
//   valueType,
//                           HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &bufferSizeL);
//   hipMallocAsync(&bufferL, bufferSizeL, stream);
//   hipsparseSpSV_analysis(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, vecTmp, vecRaux, valueType,
//                         HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, bufferL);
//   hipMemsetAsync(p_raux, 0x0, m * sizeof(ValueType), stream);
//   hipsparseSpSV_solve(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, vecTmp, vecRaux, valueType,
//                      HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL);

//   // p = raux
//   hipMemcpyAsync(p_p, p_raux, m * sizeof(ValueType), hipMemcpyDeviceToDevice, stream);

//   // ValueType r_norm;
//   // hipblasDnrm2(cublas_handle, m, p_r, 1, &r_norm);

//   // delta = r^T r
//   ValueType delta;
//   ValueType r_norm;
//   if constexpr (std::is_same<ValueType, float>::value) {
//     hipblasSdot(cublas_handle, m, p_r, 1, p_r, 1, &delta);
//     hipblasSnrm2(cublas_handle, m, p_r, 1, r_norm);
//   } else {
//     hipblasDdot(cublas_handle, m, p_r, 1, p_r, 1, &delta);
//     hipblasDnrm2(cublas_handle, m, p_r, 1, r_norm);
//   }

//   ValueType r_norm_init = r_norm;

//   for (int i = 0; i < max_iterations; ++i) {
//     // t = A p
//     hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, vecP, &zero, vecT, valueType,
//                  HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV);

//     // denom = p^T A p
//     ValueType denom;
//     if constexpr (std::is_same<ValueType, float>::value) {
//       hipblasSdot(cublas_handle, m, p_t, 1, p_p, 1, &denom);
//     } else {
//       hipblasDdot(cublas_handle, m, p_t, 1, p_p, 1, &denom);
//     }

//     // alpha = delta / denom
//     ValueType alpha = delta / denom;
//     ValueType minus_alpha = -alpha;

//     // x = x + alpha * p
//     // r = r - alpha * t
//     if constexpr (std::is_same<ValueType, float>::value) {
//       hipblasSaxpy(cublas_handle, m, &alpha, p_p, 1, p_x, 1);
//       hipblasSaxpy(cublas_handle, m, &minus_alpha, p_t, 1, p_r, 1);
//       hipblasSnrm2(cublas_handle, m, p_r, 1, r_norm);
//     } else {
//       hipblasDaxpy(cublas_handle, m, &alpha, p_p, 1, p_x, 1);
//       hipblasDaxpy(cublas_handle, m, &minus_alpha, p_t, 1, p_r, 1);
//       hipblasDnrm2(cublas_handle, m, p_r, 1, r_norm);
//     }

//     if (r_norm < r_norm_init * rtol) {
//       break;
//     }

//     hipMemsetAsync(p_tmp, 0x0, m * sizeof(ValueType), stream);
//     hipMemsetAsync(p_raux, 0x0, m * sizeof(ValueType), stream);

//     // tmp = L^-1 r
//     hipsparseSpSV_solve(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, vecR, vecTmp, valueType,
//                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL);
//     // raux = L^-T L^-1 r
//     hipsparseSpSV_solve(cusparse_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, vecTmp, vecRaux, valueType,
//                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT);

//     ValueType delta_new;
//     hipblasDdot(cublas_handle, m, p_r, 1, p_r, 1, &delta_new);

//     ValueType beta = delta_new / delta;
//     delta = delta_new;

//     // p = raux
//     hipMemcpyAsync(p_p, p_raux, m * sizeof(ValueType), hipMemcpyDeviceToDevice, stream);

//     // p = beta * p + raux
//     hipblasDaxpy(cublas_handle, m, &beta, p_p, 1, p_p, 1);
//   }

//   // r = b
//   hipMemcpyAsync(p_r, p_b, m * sizeof(ValueType), hipMemcpyDeviceToDevice, stream);

//   // r = b - Ax
//   hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one, matA, vecX, &one, vecR, valueType,
//                HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV);

//   if constexpr (std::is_same<ValueType, float>::value) {
//     hipblasSnrm2(cublas_handle, m, p_r, 1, &r_norm);
//   } else {
//     hipblasDnrm2(cublas_handle, m, p_r, 1, &r_norm);
//   }
//   printf("Final error norm = %e\n", r_norm);

//   hipsparseSpSV_destroyDescr(spsvDescrLT);
//   hipsparseSpSV_destroyDescr(spsvDescrL);

//   hipsparseDestroyDnVec(vecX);
//   hipsparseDestroyDnVec(vecR);
//   hipsparseDestroyDnVec(vecB);
//   hipsparseDestroyDnVec(vecT);
//   hipsparseDestroyDnVec(vecP);
//   hipsparseDestroyDnVec(vecRaux);
//   hipsparseDestroyDnVec(vecTmp);
//   hipsparseDestroySpMat(matL);
//   hipsparseDestroySpMat(matA);

//   hipFreeAsync(bufferL, stream);
//   hipFreeAsync(bufferLT, stream);
//   hipFreeAsync(bufferMV, stream);
//   hipFreeAsync(p_t, stream);
//   hipFreeAsync(p_p, stream);
//   hipFreeAsync(p_raux, stream);
//   hipFreeAsync(p_tmp, stream);
//   hipFreeAsync(L_values, stream);
// }

template <hipsparseOperation_t op, hipsparseFillMode_t fillmode, hipsparseDiagType_t diagtype>
static void csr_solve_tri_device_(int n, int nnz, int *offsets, int *columns, float *values, float *X, float *Y) {
  hipsparseHandle_t handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  hipsparseSetStream(handle, (hipStream_t)cuda_stream_get_current());

  float alpha = 1.f;

  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;

  hipsparseSpSVDescr_t spsvDescr;

  hipsparseCreateCsr(&matA, n, n, nnz, offsets, columns, values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

  hipsparseCreateDnVec(&vecX, n, X, HIP_R_32F);
  hipsparseCreateDnVec(&vecY, n, Y, HIP_R_32F);

  hipsparseSpSV_createDescr(&spsvDescr);

  hipsparseFillMode_t fillmode_ = fillmode;
  hipsparseDiagType_t diagtype_ = diagtype;

  hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_FILL_MODE, &fillmode_, sizeof(fillmode));
  hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype_, sizeof(diagtype));

  size_t bufferSize = 0;
  void *buffer = nullptr;
  hipsparseSpSV_bufferSize(handle, op, &alpha, matA, vecX, vecY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr,
                          &bufferSize);

  hipMalloc(&buffer, bufferSize); // preallocate?
  hipsparseSpSV_analysis(handle, op, &alpha, matA, vecX, vecY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, buffer);
  hipsparseSpSV_solve(handle, op, &alpha, matA, vecX, vecY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr);

  hipsparseDestroySpMat(matA);
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
  hipsparseSpSV_destroyDescr(spsvDescr);
  hipFree(buffer);
}

void csr_solve_lt_device(int n, int nnz, int *offsets, int *columns, float *values, float *X, float *Y) {
  csr_solve_tri_device_<HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_NON_UNIT>(
      n, nnz, offsets, columns, values, X, Y);
}
