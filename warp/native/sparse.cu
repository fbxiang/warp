#include "hip/hip_runtime.h"
#include "warp.h"
#include <hipblas.h>
#include <hipsparse.h>

namespace wp {

static hipblasHandle_t g_cublas_handle;
static hipsparseHandle_t g_cusparse_handle;

bool init_cublas() {
  hipblasStatus_t status = hipblasCreate(&g_cublas_handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUBLAS error: %d\n", status);
    return false;
  }
  return true;
}
void destroy_cublas() { hipblasDestroy(g_cublas_handle); }
void *get_cublas_handle() { return (void *)g_cublas_handle; }

bool init_cusparse() {
  hipsparseStatus_t status = hipsparseCreate(&g_cusparse_handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    fprintf(stderr, "%s\n", hipsparseGetErrorString(status));
    return false;
  }
  return true;
}
void destroy_cusparse() { hipsparseDestroy(g_cusparse_handle); }
void *get_cusparse_handle() { return (void *)g_cusparse_handle; }

} // namespace wp

// -------------------- begin helper functions --------------------

// incomplete cholesky
template <typename T>
inline hipsparseStatus_t cusparseXcsric02_bufferSize(hipsparseHandle_t handle, int m, int nnz,
                                                    const hipsparseMatDescr_t descrA, T *csrSortedValA,
                                                    const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                    csric02Info_t info, int *pBufferSizeInBytes) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsric02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                       pBufferSizeInBytes);
  } else {
    return hipsparseDcsric02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                       pBufferSizeInBytes);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsric02_analysis(hipsparseHandle_t handle, int m, int nnz,
                                                  const hipsparseMatDescr_t descrA, const T *csrSortedValA,
                                                  const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                  csric02Info_t info, hipsparseSolvePolicy_t policy, void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsric02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                     policy, pBuffer);
  } else {
    return hipsparseDcsric02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                     policy, pBuffer);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsric02(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA,
                                         T *csrSortedValA_valM, const int *csrSortedRowPtrA,
                                         const int *csrSortedColIndA, csric02Info_t info, hipsparseSolvePolicy_t policy,
                                         void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsric02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                            policy, pBuffer);
  } else {
    return hipsparseDcsric02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                            policy, pBuffer);
  }
}

// incomplete LU
template <typename T>
inline hipsparseStatus_t cusparseXcsrilu02_bufferSize(hipsparseHandle_t handle, int m, int nnz,
                                                     const hipsparseMatDescr_t descrA, T *csrSortedValA,
                                                     const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                     csrilu02Info_t info, int *pBufferSizeInBytes) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsrilu02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                        pBufferSizeInBytes);
  } else {
    return hipsparseDcsrilu02_bufferSize(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                        pBufferSizeInBytes);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsrilu02_analysis(hipsparseHandle_t handle, int m, int nnz,
                                                   const hipsparseMatDescr_t descrA, const T *csrSortedValA,
                                                   const int *csrSortedRowPtrA, const int *csrSortedColIndA,
                                                   csrilu02Info_t info, hipsparseSolvePolicy_t policy, void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsrilu02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                      policy, pBuffer);
  } else {
    return hipsparseDcsrilu02_analysis(handle, m, nnz, descrA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA, info,
                                      policy, pBuffer);
  }
}

template <typename T>
inline hipsparseStatus_t cusparseXcsrilu02(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA,
                                          T *csrSortedValA_valM, const int *csrSortedRowPtrA,
                                          const int *csrSortedColIndA, csrilu02Info_t info,
                                          hipsparseSolvePolicy_t policy, void *pBuffer) {
  if constexpr (std::is_same<T, float>::value) {
    return hipsparseScsrilu02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                             policy, pBuffer);
  } else {
    return hipsparseDcsrilu02(handle, m, nnz, descrA, csrSortedValA_valM, csrSortedRowPtrA, csrSortedColIndA, info,
                             policy, pBuffer);
  }
}

// -------------------- end helper functions --------------------

template <typename ValueType = float>
static void _csr_ichol_device(int m, int nnz, int *offsets, int *columns, ValueType *values, ValueType *L_values) {
  hipStream_t stream = (hipStream_t)cuda_stream_get_current();
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  hipblasHandle_t cublas_handle = (hipblasHandle_t)wp::get_cublas_handle();
  hipsparseSetStream(cusparse_handle, stream);
  hipblasSetStream(cublas_handle, stream);

  hipsparseMatDescr_t matM{};
  csric02Info_t infoM{};
  int bufferSizeIC{0};
  void *bufferIC;
  hipsparseCreateMatDescr(&matM);
  hipsparseSetMatIndexBase(matM, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(matM, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatFillMode(matM, HIPSPARSE_FILL_MODE_LOWER);
  hipsparseSetMatDiagType(matM, HIPSPARSE_DIAG_TYPE_NON_UNIT);
  hipsparseCreateCsric02Info(&infoM);

  // L = A
  hipMemcpyAsync(L_values, values, nnz * sizeof(ValueType), hipMemcpyDeviceToDevice);
  cusparseXcsric02_bufferSize<ValueType>(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM,
                                         &bufferSizeIC);
  hipMallocAsync(&bufferIC, bufferSizeIC, stream); // TODO preallocate
  cusparseXcsric02_analysis<ValueType>(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM,
                                       HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferIC);

  // TODO
  // int structural_zero;
  // hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &structural_zero);

  cusparseXcsric02<ValueType>(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM,
                              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferIC);

  // TODO
  // int numerical_zero;
  // hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &numerical_zero);

  hipsparseDestroyCsric02Info(infoM);
  hipsparseDestroyMatDescr(matM);
  hipFreeAsync(bufferIC, stream);
}

template <typename ValueType = float>
static void _csr_ilu_device(int m, int nnz, int *offsets, int *columns, ValueType *values, ValueType *LU_values) {
  // L has unit diagonal, U has non-unit diagonal
  hipStream_t stream = (hipStream_t)cuda_stream_get_current();
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  hipblasHandle_t cublas_handle = (hipblasHandle_t)wp::get_cublas_handle();
  hipsparseSetStream(cusparse_handle, stream);
  hipblasSetStream(cublas_handle, stream);

  hipsparseMatDescr_t matM{};
  csrilu02Info_t infoM{};
  int bufferSizeIC{0};
  void *bufferILU;
  hipsparseCreateMatDescr(&matM);
  hipsparseSetMatIndexBase(matM, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(matM, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatFillMode(matM, HIPSPARSE_FILL_MODE_LOWER);
  hipsparseSetMatDiagType(matM, HIPSPARSE_DIAG_TYPE_NON_UNIT);
  hipsparseCreateCsrilu02Info(&infoM);

  // L = A
  hipMemcpyAsync(LU_values, values, nnz * sizeof(ValueType), hipMemcpyDeviceToDevice);
  cusparseXcsrilu02_bufferSize<ValueType>(cusparse_handle, m, nnz, matM, LU_values, offsets, columns, infoM,
                                          &bufferSizeIC);
  hipMallocAsync(&bufferILU, bufferSizeIC, stream); // TODO preallocate
  cusparseXcsrilu02_analysis<ValueType>(cusparse_handle, m, nnz, matM, LU_values, offsets, columns, infoM,
                                        HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferILU);

  // TODO
  // int structural_zero;
  // hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &structural_zero);

  cusparseXcsrilu02<ValueType>(cusparse_handle, m, nnz, matM, LU_values, offsets, columns, infoM,
                               HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferILU);

  // TODO
  // int numerical_zero;
  // hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &numerical_zero);

  hipsparseDestroyCsrilu02Info(infoM);
  hipsparseDestroyMatDescr(matM);
  hipFreeAsync(bufferILU, stream);
}

void csr_ichol_device(int m, int nnz, int *offsets, int *columns, float *values, float *L_values) {
  _csr_ichol_device<float>(m, nnz, offsets, columns, values, L_values);
}

void csr_ilu_device(int m, int nnz, int *offsets, int *columns, float *values, float *LU_values) {
  _csr_ilu_device<float>(m, nnz, offsets, columns, values, LU_values);
}

/** Ax = b */
template <typename ValueType = float>
static void csr_ic_cg(int m, int nnz, int *offsets, int *columns, ValueType *values, // A
                      ValueType *p_b,                                                // RHS b
                      ValueType *p_x,                                                // initial x
                      ValueType *p_r,                                                // residual
                      int max_iterations, ValueType rtol) {
  const ValueType minus_one = -1;
  const ValueType one = 1;
  const ValueType zero = 0;
  hipsparseFillMode_t fill_lower = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseDiagType_t diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

  // set up handles
  hipStream_t stream = (hipStream_t)cuda_stream_get_current();
  hipsparseHandle_t cusparse_handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  hipblasHandle_t cublas_handle = (hipblasHandle_t)wp::get_cublas_handle();
  hipsparseSetStream(cusparse_handle, stream);
  hipblasSetStream(cublas_handle, stream);

  // set up data type
  static_assert(std::is_same<ValueType, float>::value || std::is_same<ValueType, double>::value,
                "invalid data type for csr_cg");
  constexpr hipDataType valueType = std::is_same<ValueType, float>::value ? HIP_R_32F : HIP_R_64F;

  // set up descriptors
  hipsparseSpMatDescr_t matA{};
  hipsparseSpMatDescr_t matL{};

  hipsparseCreateCsr(&matA, m, m, nnz, offsets, columns, values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, valueType);

  ValueType *L_values{};
  hipMallocAsync(&L_values, nnz * sizeof(ValueType), stream);
  hipMemcpyAsync(L_values, values, nnz * sizeof(ValueType), hipMemcpyDeviceToDevice);

  hipsparseCreateCsr(&matL, m, m, nnz, offsets, columns, L_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, valueType);
  hipsparseSpMatSetAttribute(matL, HIPSPARSE_SPMAT_FILL_MODE, &fill_lower, sizeof(fill_lower));
  hipsparseSpMatSetAttribute(matL, HIPSPARSE_SPMAT_DIAG_TYPE, &diag_non_unit, sizeof(diag_non_unit));

  // incomplete Cholesky
  hipsparseMatDescr_t matM{};
  csric02Info_t infoM{};
  int bufferSizeIC{0};
  void *bufferIC;
  hipsparseCreateMatDescr(&matM);
  hipsparseSetMatIndexBase(matM, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(matM, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatFillMode(matM, HIPSPARSE_FILL_MODE_LOWER);
  hipsparseSetMatDiagType(matM, HIPSPARSE_DIAG_TYPE_NON_UNIT);
  hipsparseCreateCsric02Info(&infoM);

  if constexpr (std::is_same<ValueType, float>::value) {
    hipsparseScsric02_bufferSize(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM, &bufferSizeIC);
  } else {
    hipsparseDcsric02_bufferSize(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM, &bufferSizeIC);
  }
  hipMallocAsync(&bufferIC, bufferSizeIC, stream); // TODO preallocate

  if constexpr (std::is_same<ValueType, float>::value) {
    hipsparseScsric02_analysis(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM,
                              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferIC);
  } else {
    hipsparseDcsric02_analysis(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM,
                              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferIC);
  }

  // TODO
  // int structural_zero;
  // hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &structural_zero);

  if constexpr (std::is_same<ValueType, float>::value) {
    hipsparseScsric02(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM, HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
                     bufferIC);
  } else {
    hipsparseDcsric02(cusparse_handle, m, nnz, matM, L_values, offsets, columns, infoM, HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
                     bufferIC);
  }

  // TODO
  // int numerical_zero;
  // hipsparseXcsric02_zeroPivot(cusparse_handle, infoM, &numerical_zero);

  hipsparseDestroyCsric02Info(infoM);
  hipsparseDestroyMatDescr(matM);
  hipFreeAsync(bufferIC, stream);

  //// CG

  // set up
  hipsparseDnVecDescr_t vecB{};
  hipsparseDnVecDescr_t vecR{};
  hipsparseDnVecDescr_t vecX{};
  hipsparseDnVecDescr_t vecTmp{};
  hipsparseDnVecDescr_t vecRaux{};
  hipsparseDnVecDescr_t vecP{};
  hipsparseDnVecDescr_t vecT{};

  ValueType *p_tmp{};
  hipMallocAsync(&p_tmp, m * sizeof(ValueType), stream);

  hipsparseCreateDnVec(&vecTmp, m, p_tmp, valueType);
  ValueType *p_raux{};
  hipMallocAsync(&p_raux, m * sizeof(ValueType), stream);
  hipsparseCreateDnVec(&vecRaux, m, p_raux, valueType);

  ValueType *p_p{};
  hipMallocAsync(&p_p, m * sizeof(ValueType), stream);
  hipsparseCreateDnVec(&vecP, m, p_p, valueType);

  ValueType *p_t{};
  hipMallocAsync(&p_t, m * sizeof(ValueType), stream);
  hipsparseCreateDnVec(&vecT, m, p_t, valueType);

  hipsparseCreateDnVec(&vecB, m, p_b, valueType);
  hipsparseCreateDnVec(&vecR, m, p_r, valueType);
  hipsparseCreateDnVec(&vecX, m, p_x, valueType);

  size_t bufferSizeMV{};
  void *bufferMV{};
  // allocate memory
  hipsparseSpMV_bufferSize(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one, matA, vecX, &one, vecB,
                          valueType, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSizeMV);
  hipMallocAsync(&bufferMV, bufferSizeMV, stream); // TODO preallocate and reuse

  // FIXME check bufferSizeMV for all vecs

  // r0 = b - A@x0
  hipMemcpyAsync(p_r, p_b, m * sizeof(ValueType), hipMemcpyDeviceToDevice, stream);
  hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one, matA, vecX, &one, vecR, valueType,
               HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV);

  // tmp = L^-T r
  size_t bufferSizeL, bufferSizeLT;
  void *bufferL, *bufferLT;
  hipsparseSpSVDescr_t spsvDescrL, spsvDescrLT;
  hipsparseSpSV_createDescr(&spsvDescrLT);

  hipsparseSpSV_bufferSize(cusparse_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, vecR, vecTmp, valueType,
                          HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT, &bufferSizeLT);
  hipMallocAsync(&bufferLT, bufferSizeLT, stream);
  hipsparseSpSV_analysis(cusparse_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, vecR, vecTmp, valueType,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT, bufferLT);
  hipMemsetAsync(p_tmp, 0x0, m * sizeof(ValueType), stream);
  hipsparseSpSV_solve(cusparse_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, vecR, vecTmp, valueType,
                     HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT);

  // raux = L^-1 L^-T r
  hipsparseSpSV_createDescr(&spsvDescrL);

  hipsparseSpSV_bufferSize(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, vecTmp, vecRaux, valueType,
                          HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &bufferSizeL);
  hipMallocAsync(&bufferL, bufferSizeL, stream);
  hipsparseSpSV_analysis(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, vecTmp, vecRaux, valueType,
                        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, bufferL);
  hipMemsetAsync(p_raux, 0x0, m * sizeof(ValueType), stream);
  hipsparseSpSV_solve(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, vecTmp, vecRaux, valueType,
                     HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL);

  // p = raux
  hipMemcpyAsync(p_p, p_raux, m * sizeof(ValueType), hipMemcpyDeviceToDevice, stream);

  // ValueType r_norm;
  // hipblasDnrm2(cublas_handle, m, p_r, 1, &r_norm);

  // delta = r^T r
  ValueType delta;
  ValueType r_norm;
  if constexpr (std::is_same<ValueType, float>::value) {
    hipblasSdot(cublas_handle, m, p_r, 1, p_r, 1, &delta);
    hipblasSnrm2(cublas_handle, m, p_r, 1, r_norm);
  } else {
    hipblasDdot(cublas_handle, m, p_r, 1, p_r, 1, &delta);
    hipblasDnrm2(cublas_handle, m, p_r, 1, r_norm);
  }

  ValueType r_norm_init = r_norm;

  for (int i = 0; i < max_iterations; ++i) {
    // t = A p
    hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, vecP, &zero, vecT, valueType,
                 HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV);

    // denom = p^T A p
    ValueType denom;
    if constexpr (std::is_same<ValueType, float>::value) {
      hipblasSdot(cublas_handle, m, p_t, 1, p_p, 1, &denom);
    } else {
      hipblasDdot(cublas_handle, m, p_t, 1, p_p, 1, &denom);
    }

    // alpha = delta / denom
    ValueType alpha = delta / denom;
    ValueType minus_alpha = -alpha;

    // x = x + alpha * p
    // r = r - alpha * t
    if constexpr (std::is_same<ValueType, float>::value) {
      hipblasSaxpy(cublas_handle, m, &alpha, p_p, 1, p_x, 1);
      hipblasSaxpy(cublas_handle, m, &minus_alpha, p_t, 1, p_r, 1);
      hipblasSnrm2(cublas_handle, m, p_r, 1, r_norm);
    } else {
      hipblasDaxpy(cublas_handle, m, &alpha, p_p, 1, p_x, 1);
      hipblasDaxpy(cublas_handle, m, &minus_alpha, p_t, 1, p_r, 1);
      hipblasDnrm2(cublas_handle, m, p_r, 1, r_norm);
    }

    if (r_norm < r_norm_init * rtol) {
      break;
    }

    hipMemsetAsync(p_tmp, 0x0, m * sizeof(ValueType), stream);
    hipMemsetAsync(p_raux, 0x0, m * sizeof(ValueType), stream);

    // tmp = L^-1 r
    hipsparseSpSV_solve(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, vecR, vecTmp, valueType,
                       HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL);
    // raux = L^-T L^-1 r
    hipsparseSpSV_solve(cusparse_handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, vecTmp, vecRaux, valueType,
                       HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLT);

    ValueType delta_new;
    hipblasDdot(cublas_handle, m, p_r, 1, p_r, 1, &delta_new);

    ValueType beta = delta_new / delta;
    delta = delta_new;

    // p = raux
    hipMemcpyAsync(p_p, p_raux, m * sizeof(ValueType), hipMemcpyDeviceToDevice, stream);

    // p = beta * p + raux
    hipblasDaxpy(cublas_handle, m, &beta, p_p, 1, p_p, 1);
  }

  // r = b
  hipMemcpyAsync(p_r, p_b, m * sizeof(ValueType), hipMemcpyDeviceToDevice, stream);

  // r = b - Ax
  hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus_one, matA, vecX, &one, vecR, valueType,
               HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV);

  if constexpr (std::is_same<ValueType, float>::value) {
    hipblasSnrm2(cublas_handle, m, p_r, 1, &r_norm);
  } else {
    hipblasDnrm2(cublas_handle, m, p_r, 1, &r_norm);
  }
  printf("Final error norm = %e\n", r_norm);

  hipsparseSpSV_destroyDescr(spsvDescrLT);
  hipsparseSpSV_destroyDescr(spsvDescrL);

  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecR);
  hipsparseDestroyDnVec(vecB);
  hipsparseDestroyDnVec(vecT);
  hipsparseDestroyDnVec(vecP);
  hipsparseDestroyDnVec(vecRaux);
  hipsparseDestroyDnVec(vecTmp);
  hipsparseDestroySpMat(matL);
  hipsparseDestroySpMat(matA);

  hipFreeAsync(bufferL, stream);
  hipFreeAsync(bufferLT, stream);
  hipFreeAsync(bufferMV, stream);
  hipFreeAsync(p_t, stream);
  hipFreeAsync(p_p, stream);
  hipFreeAsync(p_raux, stream);
  hipFreeAsync(p_tmp, stream);
  hipFreeAsync(L_values, stream);
}

template <hipsparseOperation_t op, hipsparseFillMode_t fillmode, hipsparseDiagType_t diagtype>
static void csr_solve_tri_device_(int n, int nnz, int *offsets, int *columns, float *values, float *X, float *Y) {
  hipsparseHandle_t handle = (hipsparseHandle_t)wp::get_cusparse_handle();
  hipsparseSetStream(handle, (hipStream_t)cuda_stream_get_current());

  float alpha = 1.f;

  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;

  hipsparseSpSVDescr_t spsvDescr;

  hipsparseCreateCsr(&matA, n, n, nnz, offsets, columns, values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

  hipsparseCreateDnVec(&vecX, n, X, HIP_R_32F);
  hipsparseCreateDnVec(&vecY, n, Y, HIP_R_32F);

  hipsparseSpSV_createDescr(&spsvDescr);

  hipsparseFillMode_t fillmode_ = fillmode;
  hipsparseDiagType_t diagtype_ = diagtype;

  hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_FILL_MODE, &fillmode_, sizeof(fillmode));
  hipsparseSpMatSetAttribute(matA, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype_, sizeof(diagtype));

  size_t bufferSize = 0;
  void *buffer = nullptr;
  hipsparseSpSV_bufferSize(handle, op, &alpha, matA, vecX, vecY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr,
                          &bufferSize);

  hipMalloc(&buffer, bufferSize); // preallocate?
  hipsparseSpSV_analysis(handle, op, &alpha, matA, vecX, vecY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, buffer);
  hipsparseSpSV_solve(handle, op, &alpha, matA, vecX, vecY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr);

  hipsparseDestroySpMat(matA);
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
  hipsparseSpSV_destroyDescr(spsvDescr);
  hipFree(buffer);
}

void csr_solve_lt_device(int n, int nnz, int *offsets, int *columns, float *values, float *X, float *Y) {
  csr_solve_tri_device_<HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_NON_UNIT>(
      n, nnz, offsets, columns, values, X, Y);
}
